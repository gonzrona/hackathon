#include "hip/hip_runtime.h"
#include <stdio.h>

#include "cuda_helper.h"
#include "cuda_kernels.h"

__device__ hipDoubleComplex ComplexScale( hipDoubleComplex const &a, double const &scale ) {
    hipDoubleComplex c;
    c.x = a.x * scale;
    c.y = a.y * scale;
    return ( c );
}

// for ( j = 0; j < Ny; j++ ) {
//     for ( i = 0; i < Nx; i++ ) {
//         in1[i + 1 + j * NR] = creal( rhs[( j * Nx + i ) + ( l * Nxy )] );
//     }
// }
__global__ void __launch_bounds__( 256 ) load_1st_DST( const int l,
                                                       const int NR,
                                                       const int Nx,
                                                       const int Ny,
                                                       const hipDoubleComplex *__restrict__ rhs,
                                                       double *__restrict__ in ) {
    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int strideX { static_cast<int>( blockDim.x * gridDim.x ) };

    const int ty { static_cast<int>( blockIdx.y * blockDim.y + threadIdx.y ) };
    const int strideY { static_cast<int>( blockDim.y * gridDim.y ) };

    for ( int tidY = ty; tidY < Ny; tidY += strideY ) {
        for ( int tidX = tx; tidX < Nx; tidX += strideX ) {
            in[tidY * NR + tidX + 1]                   = rhs[( tidY * Nx + tidX ) + ( l * Nx * Ny )].x;
            in[( NR * Ny ) + ( tidY * NR + tidX + 1 )] = rhs[( tidY * Nx + tidX ) + ( l * Nx * Ny )].y;
        }
    }
}

// for ( j = 0; j < Ny; j++ ) {
//     for ( i = 0; i < Nx; i++ ) {
//         in2[j + 1 + i * NR] = out1[i + 1 + j * NC].y;
//     }
// }
__global__ void __launch_bounds__( 256 ) load_2st_DST( const int l,
                                                       const int NR,
                                                       const int NC,
                                                       const int Nx,
                                                       const int Ny,
                                                       const hipDoubleComplex *__restrict__ out,
                                                       double *__restrict__ in ) {
    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int strideX { static_cast<int>( blockDim.x * gridDim.x ) };

    const int ty { static_cast<int>( blockIdx.y * blockDim.y + threadIdx.y ) };
    const int strideY { static_cast<int>( blockDim.y * gridDim.y ) };

    for ( int tidY = ty; tidY < Ny; tidY += strideY ) {
        for ( int tidX = tx; tidX < Nx; tidX += strideX ) {
            in[tidX * NR + tidY + 1]                   = out[tidY * NC + tidX + 1].y;
            in[( NR * Ny ) + ( tidX * NR + tidY + 1 )] = out[( NC * Ny ) + ( tidY * NC + tidX + 1 )].y;
        }
    }
}

// for ( j = 0; j < Ny; j++ ) {
//     for ( i = 0; i < Nx; i++ ) {
//         rhat[( j * Nx + i ) + ( l * Nxy )].x = coef * out2[j + 1 + i * NC].y;
//         rhat[( j * Nx + i ) + ( l * Nxy )].y = coef * out2[( NC * Ny ) + (j + 1 + i * NC)].y;
//     }
// }
__global__ void __launch_bounds__( 256 ) store_1st_DST( const int    l,
                                                        const int    NR,
                                                        const int    NC,
                                                        const int    Nx,
                                                        const int    Ny,
                                                        const double coef,
                                                        const hipDoubleComplex *__restrict__ out,
                                                        hipDoubleComplex *__restrict__ d_rhat ) {

    const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
    const int strideX { static_cast<int>( blockDim.x * gridDim.x ) };

    const int ty { static_cast<int>( blockIdx.y * blockDim.y + threadIdx.y ) };
    const int strideY { static_cast<int>( blockDim.y * gridDim.y ) };

    for ( int tidY = ty; tidY < Ny; tidY += strideY ) {
        for ( int tidX = tx; tidX < Nx; tidX += strideX ) {
            d_rhat[( Nx * tidY + tidX ) + ( l * Nx * Ny )].x = coef * out[tidX * NC + tidY + 1].y;
            d_rhat[( Nx * tidY + tidX ) + ( l * Nx * Ny )].y = coef * out[( NC * Ny ) + ( tidX * NC + tidY + 1 )].y;
        }
    }
}

// // #pragma omp for
// //   for (j = 0; j < Ny; j++) {
// //     for (i = 0; i < dst.Nx; i++) {
// //       in[(j * N) + i + 1] = creal(xhat[j + i * Ny]);
// //     }
// //     for (i = 0; i < dst.Nx; i++) {
// //       in2[(j * N) + i + 1] = cimag(xhat[j + i * Ny]);
// //     }
// //   }
// __global__ void __launch_bounds__( 256 ) load_2st_DST( const int N,
//                                                        const int Nx,
//                                                        const int Ny,
//                                                        const hipDoubleComplex *__restrict__ xhat,
//                                                        double *__restrict__ in ) {
//     const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
//     const int strideX { static_cast<int>( blockDim.x * gridDim.x ) };

//     const int ty { static_cast<int>( blockIdx.y * blockDim.y + threadIdx.y ) };
//     const int strideY { static_cast<int>( blockDim.y * gridDim.y ) };

//     for ( int tidY = ty; tidY < Ny; tidY += strideY ) {
//         for ( int tidX = tx; tidX < Nx; tidX += strideX ) {
//             in[tidY * N + tidX + 1]                  = xhat[tidY + tidX * Ny].x;
//             in[( N * Ny ) + ( tidY * N + tidX + 1 )] = xhat[tidY + tidX * Ny].y;
//         }
//     }
// }

// // #pragma omp for
// //   for (j = 0; j < Ny; j++) {
// //     my = j * Nx;

// //     for (i = 0; i < dst.Nx; i++) {
// //       sol[i + my] = dst.coef * (-cimag(out[(j * NC) + i + 1]) -
// //                                 I * cimag(out2[(j * NC) + i + 1]));
// //     }
// //   }
// __global__ void __launch_bounds__( 256 ) store_2st_DST( const int    N,
//                                                         const int    Nx,
//                                                         const int    Ny,
//                                                         const int    NC,
//                                                         const double coef,
//                                                         const hipDoubleComplex *__restrict__ out,
//                                                         hipDoubleComplex *__restrict__ d_sol ) {
//     const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
//     const int strideX { static_cast<int>( blockDim.x * gridDim.x ) };

//     const int ty { static_cast<int>( blockIdx.y * blockDim.y + threadIdx.y ) };
//     const int strideY { static_cast<int>( blockDim.y * gridDim.y ) };

//     for ( int tidY = ty; tidY < Ny; tidY += strideY ) {
//         for ( int tidX = tx; tidX < Nx; tidX += strideX ) {
//             d_sol[Nx * tidY + tidX].x = coef * -out[tidY * NC + tidX + 1].y;
//             d_sol[Nx * tidY + tidX].y = coef * -out[( NC * Ny ) + ( tidY * NC + tidX + 1 )].y;
//         }
//     }
// }

// // for (i = 0; i < Nx; i++) {
// //   y[0] = rhat[i];
// //   mx = i * Ny;
// //   for (j = 1; j < Ny; j++) {
// //     y[j] = rhat[ind(i, j, Nx)] - sys.L[j + mx] * y[j - 1];
// //   }
// //   xhat[Ny - 1 + mx] = y[Ny - 1] / sys.U[Ny - 1 + mx];
// //   for (j = Ny - 2; j >= 0; j--) {
// //     xhat[j + mx] = (y[j] - sys.Up[j + mx] * xhat[j + 1 + mx]) / sys.U[j + mx];
// //   }
// // }
// __global__ void __launch_bounds__( 256 ) middle_stuff_DST( const int N,
//                                                            const int Nx,
//                                                            const int Ny,
//                                                            const hipDoubleComplex *__restrict__ d_SysU,
//                                                            const hipDoubleComplex *__restrict__ d_SysL,
//                                                            const hipDoubleComplex *__restrict__ d_SysUp,
//                                                            const hipDoubleComplex *__restrict__ d_rhat,
//                                                            hipDoubleComplex *__restrict__ d_xhat,
//                                                            hipDoubleComplex *__restrict__ d_y ) {
//     const int tx { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };
//     const int strideX { static_cast<int>( blockDim.x * gridDim.x ) };

//     for ( int tidX = tx; tidX < Nx; tidX += strideX ) {
//         int mx = tidX * Ny;

//         d_y[tidX] = d_rhat[tidX];

//         for ( int j = 1; j < Ny; j++ ) {
//             d_y[j * Ny + tidX] =
//                 // d_rhat[ind(tidX, j, Nx)] - sys.L[j + mx] * d_y[(j - 1) + mx];
//                 hipCsub( d_rhat[ind( tidX, j, Nx )], hipCmul( d_SysL[j + mx], d_y[( j - 1 ) * Ny + tidX] ) );
//         }

//         d_xhat[Ny - 1 + mx] = hipCdiv( d_y[( Ny - 1 ) * Ny + tidX], d_SysU[Ny - 1 + mx] );
//         for ( int j = Ny - 2; j >= 0; j-- ) {
//             d_xhat[j + mx] =
//                 // (d_y[j + mx] - sys.Up[j + mx] * d_xhat[j + 1 + mx]) / sys.U[j +
//                 // mx];
//                 hipCdiv( hipCsub( d_y[j * Ny + tidX], hipCmul( d_SysUp[j + mx], d_xhat[j + 1 + mx] ) ), d_SysU[j + mx]
//                 );
//         }
//     }
// }

// // #pragma omp for
// //   for (j = 0; j < Ny; j++) {
// //     my = j * Nx;

// //     for (i = 0; i < dst.Nx; i++) {
// //       rhat[i + my] = dst.coef * (-cimag(out[(j * NC) + i + 1]) -
// //                                  I * cimag(out2[(j * NC) + i + 1]));
// //     }
// //   }

// // for (i = 0; i < Nx; i++) {
// //   y[0] = rhat[i];
// //   mx = i * Ny;
// //   for (j = 1; j < Ny; j++) {
// //     y[j] = rhat[ind(i, j, Nx)] - sys.L[j + mx] * y[j - 1];
// //   }
// //   xhat[Ny - 1 + mx] = y[Ny - 1] / sys.U[Ny - 1 + mx];
// //   for (j = Ny - 2; j >= 0; j--) {
// //     xhat[j + mx] = (y[j] - sys.Up[j + mx] * xhat[j + 1 + mx]) / sys.U[j + mx];
// //   }
// // }

// // #pragma omp for
// //   for (j = 0; j < Ny; j++) {
// //     for (i = 0; i < dst.Nx; i++) {
// //       in[(j * N) + i + 1] = creal(xhat[j + i * Ny]);
// //     }
// //     for (i = 0; i < dst.Nx; i++) {
// //       in2[(j * N) + i + 1] = cimag(xhat[j + i * Ny]);
// //     }
// //   }
// __global__ void __launch_bounds__( 64 ) middle_stuff_ls_DST( const int    N,
//                                                              const int    Nx,
//                                                              const int    Ny,
//                                                              const int    NC,
//                                                              const double coef,
//                                                              const hipDoubleComplex *__restrict__ out,
//                                                              const hipDoubleComplex *__restrict__ d_SysU,
//                                                              const hipDoubleComplex *__restrict__ d_SysL,
//                                                              const hipDoubleComplex *__restrict__ d_SysUp,
//                                                              hipDoubleComplex *__restrict__ d_y,
//                                                              double *__restrict__ in ) {

//     const int tidX { static_cast<int>( blockIdx.x * blockDim.x + threadIdx.x ) };

//     hipDoubleComplex temp {};

//     if ( tidX < Nx ) {
//         int mx = Ny * tidX;

//         temp      = make_hipDoubleComplex( -out[tidX + 1].y, -out[( NC * Ny ) + tidX + 1].y );
//         temp      = ComplexScale( temp, coef );
//         d_y[tidX] = temp;

// #pragma unroll 8
//         for ( int j = 1; j < Ny; j++ ) {
//             hipDoubleComplex temp2 = hipCmul( d_SysL[mx + j], d_y[( j - 1 ) * Ny + tidX] );
//             temp = make_hipDoubleComplex( -out[j * NC + tidX + 1].y, -out[( NC * Ny ) + ( j * NC + tidX + 1 )].y );
//             temp = ComplexScale( temp, coef );
//             d_y[j * Ny + tidX] = hipCsub( temp, temp2 );
//         }

//         temp = hipCdiv( d_y[( Ny - 1 ) * Ny + tidX], d_SysU[mx + ( Ny - 1 )] );

//         in[( Ny - 1 ) * N + tidX + 1]                  = temp.x;
//         in[( N * Ny ) + ( ( Ny - 1 ) * N + tidX + 1 )] = temp.y;
// #pragma unroll 4
//         for ( int j = Ny - 2; j >= 0; j-- ) {
//             hipDoubleComplex temp2 =
//                 hipCdiv( hipCsub( d_y[j * Ny + tidX], hipCmul( d_SysUp[mx + j], temp ) ), d_SysU[mx + j] );
//             in[j * N + tidX + 1]                  = temp2.x;
//             in[( N * Ny ) + ( j * N + tidX + 1 )] = temp2.y;
//             temp                                  = temp2;
//         }
//     }
// }

void load_1st_DST_wrapper( const hipStream_t stream,
                           int                l,
                           int                Nx,
                           int                Ny,
                           int                NR,
                           hipDoubleComplex *  d_rhs,
                           double *           in ) {

    // int N = 2 * Nx + 2;

    int numSMs;
    CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );

    dim3 threadPerBlock { 16, 16 };
    dim3 blocksPerGrid( numSMs, numSMs );

    void *args[] { &l, &NR, &Nx, &Ny, &d_rhs, &in };

    CUDA_RT_CALL( hipLaunchKernel( ( void * )( &load_1st_DST ), blocksPerGrid, threadPerBlock, args, 0, stream ) );

    CUDA_RT_CALL( hipPeekAtLastError( ) );
}

void load_2st_DST_wrapper( const hipStream_t stream,
                           int                l,
                           int                Nx,
                           int                Ny,
                           int                NR,
                           int                NC,
                           hipDoubleComplex *  d_rhs,
                           double *           in ) {

    // int N = 2 * Nx + 2;

    int numSMs;
    CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );

    dim3 threadPerBlock { 16, 16 };
    dim3 blocksPerGrid( numSMs, numSMs );

    void *args[] { &l, &NR, &NC, &Nx, &Ny, &d_rhs, &in };

    CUDA_RT_CALL( hipLaunchKernel( ( void * )( &load_2st_DST ), blocksPerGrid, threadPerBlock, args, 0, stream ) );

    CUDA_RT_CALL( hipPeekAtLastError( ) );
}

void store_1st_DST_wrapper( const hipStream_t stream,
                            int                l,
                            int                Nx,
                            int                Ny,
                            int                NR,
                            int                NC,
                            hipDoubleComplex *  out,
                            hipDoubleComplex *  d_rhat ) {

    // int Nx = sys.lat.Nx, Ny = sys.lat.Ny;
    // int N = 2 * Nx + 2, NC = ( N / 2 ) + 1;

    int numSMs;
    CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );

    dim3 threadPerBlock { 16, 16 };
    dim3 blocksPerGrid( numSMs, numSMs );

    double coef = 2.0 / sqrt( Nx + 1 ) / sqrt( Ny + 1 );

    void *args[] { &l, &NR, &NC, &Nx, &Ny, &coef, &out, &d_rhat };

    CUDA_RT_CALL( hipLaunchKernel( ( void * )( &store_1st_DST ), blocksPerGrid, threadPerBlock, args, 0, stream ) );

    CUDA_RT_CALL( hipPeekAtLastError( ) );
}

// void load_2st_DST_wrapper( const hipStream_t stream,
//                            const System       sys,
//                            //    const DSTN             dst,
//                            const hipDoubleComplex *d_xhat,
//                            double *               in ) {

//     int Nx = sys.lat.Nx, Ny = sys.lat.Ny;
//     int N = 2 * Nx + 2;

//     int numSMs;
//     CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );

//     dim3 threadPerBlock { 16, 16 };
//     dim3 blocksPerGrid( numSMs, numSMs );

//     void *args[] { &N, &Nx, &Ny, &d_xhat, &in };

//     CUDA_RT_CALL( hipLaunchKernel( ( void * )( &load_2st_DST ), blocksPerGrid, threadPerBlock, args, 0, stream ) );

//     CUDA_RT_CALL( hipPeekAtLastError( ) );
// }

// void store_2st_DST_wrapper( const hipStream_t stream,
//                             const System       sys,
//                             // const DSTN             dst,
//                             const hipDoubleComplex *out,
//                             hipDoubleComplex *      d_sol ) {

//     int Nx = sys.lat.Nx, Ny = sys.lat.Ny;
//     int N = 2 * Nx + 2, NC = ( N / 2 ) + 1;

//     int numSMs;
//     CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );

//     dim3 threadPerBlock { 16, 16 };
//     dim3 blocksPerGrid( numSMs, numSMs );

//     double coef = 2.0 / sqrt( Nx + 1 ) / sqrt( Ny + 1 );

//     void *args[] { &N, &Nx, &Ny, &NC, &coef, &out, &d_sol };

//     CUDA_RT_CALL( hipLaunchKernel( ( void * )( &store_2st_DST ), blocksPerGrid, threadPerBlock, args, 0, stream ) );

//     CUDA_RT_CALL( hipPeekAtLastError( ) );
// }

// void middle_stuff_DST_wrapper( const hipStream_t     stream,
//                                System                 sys,
//                                const hipDoubleComplex *d_rhat,
//                                hipDoubleComplex *      d_xhat,
//                                hipDoubleComplex *      d_y ) {

//     int Nx = sys.lat.Nx, Ny = sys.lat.Ny;
//     int N = 2 * Nx + 2;

//     int numSMs;
//     CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );

//     int threadPerBlock { 64 };
//     int blocksPerGrid( numSMs );

//     void *args[] { &N, &Nx, &Ny, &sys.U, &sys.L, &sys.Up, &d_rhat, &d_xhat, &d_y };

//     CUDA_RT_CALL( hipLaunchKernel( ( void * )( &middle_stuff_DST ), blocksPerGrid, threadPerBlock, args, 0, stream )
//     );

//     CUDA_RT_CALL( hipPeekAtLastError( ) );
// }

// void middle_stuff_ls_DST_wrapper( const hipStream_t stream,
//                                   System             sys,
//                                   //   const DSTN             dst,
//                                   const hipDoubleComplex *out,
//                                   double *               in,
//                                   hipDoubleComplex *      d_y ) {

//     int Nx = sys.lat.Nx, Ny = sys.lat.Ny;
//     int N = 2 * Nx + 2, NC = ( N / 2 ) + 1;

//     double coef = 2.0 / sqrt( Nx + 1 ) / sqrt( Ny + 1 );

//     int numSMs;
//     CUDA_RT_CALL( hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, 0 ) );

//     int threadPerBlock { 64 };
//     int blocksPerGrid { ( N + threadPerBlock - 1 ) / threadPerBlock };

//     void *args[] { &N, &Nx, &Ny, &NC, &coef, &out, &sys.U, &sys.L, &sys.Up, &d_y, &in };

//     CUDA_RT_CALL(
//         hipLaunchKernel( ( void * )( &middle_stuff_ls_DST ), blocksPerGrid, threadPerBlock, args, 0, stream ) );

//     CUDA_RT_CALL( hipPeekAtLastError( ) );
// }